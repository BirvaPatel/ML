#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <hiprand.h>
#include <cassert>
#include <cmath>
#include <ctime>
#include <iostream>
#include <vector>
#include "addvec.h"

int matmul(const int M, const int N, const int K, const int iter) {

    // random number generator using Pseudo 
    hiprandGenerator_t prng;
    hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);

    // seed setting
    hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long)clock());

    // calculation of gpu runtime using cuda event creator.
    float elapsed = 0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    // loop to generate random A and C for given number of iteration.
    for (int n = 1; n <= iter; n++) {

        // Pre-calculate the size of our matrices
        const size_t bytes_a = M * K * sizeof(float);
        const size_t bytes_b = K * N * sizeof(float);
        const size_t bytes_c = M * N * sizeof(float);

        // Allocate device memory
        float* d_a, * d_c;
        float* d_b;
        hipMalloc(&d_a, bytes_a);
        hipMalloc(&d_b, bytes_b);
        hipMalloc(&d_c, bytes_c);

        // Filling matrix A with random numbers 
        hiprandGenerateUniform(prng, d_a, M * K);
        // Filling matrix B with random numbers 
        hiprandGenerateUniform(prng, d_b, K * M);

        // cuBLAS handle
        hipblasHandle_t handle;
        hipblasCreate(&handle);

        // Scalaing factors
        float alpha = 1.0f;
        float beta = 0.0f;

        // matrix calculation using cublas sgemm.
        hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, &alpha, d_a, M, d_b, K, &beta, d_c, M);

        // Free our memory
        hipFree(d_a);
        hipFree(d_b);
        hipFree(d_c);
    }
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    printf("time in gpu : %.2f ms\ ", elapsed);
    return 0;
}
int main() {
    // Condition -1 but with changing in matrix size and number of iterations.
    const int M = 5000;
    const int N = 5000;
    const int K = 4000;
    const int iter = 200;

    matmul(M, N, K, iter);
    std::cout << "Condition COMPLETED SUCCESSFULLY\n";
    return 0;
}